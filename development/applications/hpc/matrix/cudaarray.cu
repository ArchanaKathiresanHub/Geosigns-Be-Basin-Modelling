#include "cudaarray.h"

#include "cudaexception.h"

namespace hpc
{
 
  
CudaArray<void>
  :: CudaArray( const CudaArray & other)
  : m_size(0)
  , m_array(0)
{
  this->resize( other.size() );

  try
  {
    hipMemcpy( this->m_array, other.m_array, m_size, hipMemcpyDeviceToDevice);
    CudaException::check();
  }
  catch(...)
  {
    clear();
    throw;
  }
}

void
CudaArray<void>
  ::  resize(SizeType size)
{
  clear();

  hipMalloc( &m_array, size);
  CudaException::check();
  m_size = size;
}

void
CudaArray<void>
  :: clear()
{
  if (m_array)
  {
    hipFree(m_array);
    CudaException::check();
    m_size = 0;
    m_array = 0;
  }
}

void
CudaArray<void>
  :: getData(void * dst, SizeType srcOffset, SizeType length) const
{
  hipMemcpy(dst, static_cast<const char *>(m_array) + srcOffset, length
    , hipMemcpyDeviceToHost);
  CudaException::check();
  hipDeviceSynchronize();
  CudaException::check();
}

void
CudaArray<void>
  :: setData(const void * src, SizeType dstOffset, SizeType length) 
{
  hipMemcpy(static_cast<char *>(m_array) + dstOffset, src, length
    , hipMemcpyHostToDevice);
  CudaException::check();
  hipDeviceSynchronize();
  CudaException::check();
}


}


