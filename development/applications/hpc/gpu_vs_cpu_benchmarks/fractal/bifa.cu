/********************************************************************
  Benchmark program that measures the number of GigaFLOPS/sec
  an NVidia CUDA enabled GPU can perform while computing a recursion
  for several input parameters in parallel. 

  The recursion
      x_{n+1} = a * x_n * (1 - x_n)
  shows 'irregular' but bounded behaviour for a \in [1,4) and x_0 \in (0,1).
  Therefore an optimizing compiler cannot take any shortcuts. The resulting
  performance measurements are therefore trustworthy.
**/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <unistd.h>


const int THREADS=32;

double getTime()
{
  struct timeval t;
  gettimeofday(&t, NULL);

  return t.tv_sec + 1.0e-6 * t.tv_usec;
}

void c(hipError_t errorcode)
{
  if (errorcode != hipSuccess)
    fprintf(stderr, "CUDA ERROR = %s\n", hipGetErrorString(errorcode));
}

__global__ void iteration(int N, float amax, float amin, float xmax, float xmin, float * xs)
{
  int i;

  float a = amin + blockIdx.x * (amax - amin) / gridDim.x;
  float x = xmin + threadIdx.x * (xmax - xmin) / blockDim.x;

  for (i = 0; i < N; ++i)
  {
    x = a * x * (1 - x);
  }

  xs[blockIdx.x * blockDim.x + threadIdx.x] = x;
}

int main(int argc, char ** argv)
{
  hipGetLastError(); // clear cuda errors;

  // parameters
  const float amin = 1.0;
  const float amax = 3.99;
  const float x0min = 0.01; 
  const float x0max = 0.99;

  const int BLOCKS= 1<<13; 
  const int N = (1l << 35) / (THREADS * BLOCKS);

  // allocate output array
  float * h_xs, * d_xs;
  c( hipMalloc(&d_xs, sizeof(float)*BLOCKS * THREADS));
  c( hipMemset(d_xs, 0, sizeof(float)*BLOCKS*THREADS));

  h_xs = (float * ) malloc(sizeof(float)*BLOCKS*THREADS);
  memset( h_xs, 0, sizeof(float)*BLOCKS*THREADS);
  
  // check input
  for (int i = 0; i < BLOCKS*THREADS; ++i)
    if (h_xs[i] != 0.0)
    {
      printf("Error in validation. xs[%d] = %f\n", i, h_xs[i]);
      break; 
    }


  // run kernel
  double t0 = getTime();
  iteration<<<BLOCKS, THREADS>>>(N, amax, amin, x0max, x0min, d_xs);
  c( hipGetLastError() );

  c( hipMemcpy( h_xs, d_xs, sizeof(float)*BLOCKS*THREADS, hipMemcpyDeviceToHost) );
  c( hipDeviceSynchronize() );
  double t1 = getTime();

  // check output
  for (int i = 0; i < BLOCKS*THREADS; ++i)
    if (h_xs[i] <= 0.0 || h_xs[i] > 1.0)
    {
      printf("Error in validation. xs[%d] = %f\n", i, h_xs[i]);
      break; 
    }

  double flops = 3.0 * N * BLOCKS * THREADS / (t1 - t0);

  printf("Time: %f, GFLOPS/s = %f\n", (t1 - t0), flops * 1e-9);

  return 0;
}



// vim: ft=cpp
