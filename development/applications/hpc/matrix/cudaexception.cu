#include "cudaexception.h"


namespace hpc
{

CudaException
  :: CudaException()
{
  (*this) << "CUDA Error";
}
 
void
CudaException
  :: check()
{
  hipError_t e = hipGetLastError();
  if (e != hipSuccess)
    throw CudaException() << ": " << hipGetErrorString(e) ;
}

void
CudaException
  :: check(hipError_t e)
{
  if (e != hipSuccess)
    throw CudaException() << ": " << hipGetErrorString(e);
}

CuSparseException
  :: CuSparseException()
{
  (*this) << "CUSPARSE Library Error";
}

void
CuSparseException
  :: check(hipsparseStatus_t e)
{
  switch(e)
  {
    case HIPSPARSE_STATUS_SUCCESS:
      break;
    
    case HIPSPARSE_STATUS_NOT_INITIALIZED:
      throw CuSparseException() << ": Library wasn't initialized.";
    case HIPSPARSE_STATUS_ALLOC_FAILED:
      throw CuSparseException() << ": Could not allocated resource.";
    case HIPSPARSE_STATUS_INVALID_VALUE:
      throw CuSparseException() << ": Invalid parameter";
    case HIPSPARSE_STATUS_ARCH_MISMATCH:
      throw CuSparseException() << ": Device architecture mismatch";
    case HIPSPARSE_STATUS_MAPPING_ERROR:
      throw CuSparseException() << ": Memory error";
    case HIPSPARSE_STATUS_EXECUTION_FAILED:
      throw CuSparseException() << ": GPU program failed to execute";
    case HIPSPARSE_STATUS_INTERNAL_ERROR:
      throw CuSparseException() << ": Internal error";
    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
      throw CuSparseException() << ": Matrix type is not supported.";

    default:
      throw CuSparseException() << ": Unknown error";
  }
}

}
