/********************************************************************
  Simple benchmark program that measures the number of GigaFLOPS/sec
  an NVidia CUDA enabled GPU can perform while doing a SAXPY vector
  operation, i.e. it computes: y = alpha * x + y, where alpha is a
  scalar and x and y are vectors.

  To get an optimistic figure the saxpy is repeated ITER1 times. This
  increases the number of FLOPS per read/written bytes from/to memory.

  Note: The 'nans' in the output are for certain combinations of
     of 'threads' and 'blocks' it skipped.
**/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
#include <unistd.h>


double getTime()
{
  struct timeval t;
  gettimeofday(&t, NULL);

  return t.tv_sec + 1.0e-6 * t.tv_usec;
}

void c(hipError_t errorcode)
{
  if (errorcode != hipSuccess)
    fprintf(stderr, "CUDA ERROR = %s\n", hipGetErrorString(errorcode));
}

const int ITER1=1<<10;
const int ITER2=1<<3;

__global__ void saxpy( int n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  for (int j = 1; j <= ITER1; ++j)
    y[i] = y[i] + j * a * x[i];
}


double measureSaxpy( const int N, const int THREADS)
{
  if (N / THREADS >= 1 << 16 || THREADS > N || THREADS > 1 << 9)
    return NAN;

  float *x, *y, a;
  float *d_x, *d_y;

  // allocate memory
  x = (float *) malloc(sizeof(float)*N);
  y = (float *) malloc(sizeof(float)*N);

  c( hipMalloc(&d_x, sizeof(float)*N));
  c( hipMalloc(&d_y, sizeof(float)*N));


  // initialize memory
  a = 2.0/3.0;
  for (int i = 0; i < N; ++i)
  {
    x[i] = rand() / (RAND_MAX + 1.0);
    y[i] = rand() / (RAND_MAX + 1.0);
  }

  // start timer
  hipDeviceSynchronize();
  double t0 = getTime();

  // copy to device
  c( hipMemcpy(d_x, x, sizeof(float)*N, hipMemcpyHostToDevice) );
  c( hipMemcpy(d_y, y, sizeof(float)*N, hipMemcpyHostToDevice) );

  // run kernel
  for (int i = 0; i < ITER2; ++i)
  {
    saxpy<<< N/THREADS, THREADS>>>(N, a, d_x, d_y);
    hipError_t error = hipGetLastError();
    if ( error != hipSuccess)
    {
      fprintf(stderr, "Kernel lanch failure: %s\n", hipGetErrorString(error));
      fprintf(stderr, "Kernel arguments were: BLOCKS=%d, THREADS=%d\n", N/THREADS, THREADS);
      break;
    }
  }

  // copy result back
  c( hipMemcpy(y, d_y, sizeof(float)*N, hipMemcpyDeviceToHost) );

  // synchronize and stop timer
  hipDeviceSynchronize();
  double t3 = getTime();

  // print some statistic based on x & y;
  double sum = 0.0;
  for (int i = 0 ; i < N; ++i)
    sum += y[i];
  double avg= sum / N;

  const double epsilon = 0.1;
  const double expected = (0.5 + ITER2 * (0.5 * (ITER1+1) * ITER1) * 0.5 * a);
  if (fabs( avg - expected ) / expected > epsilon)
    fprintf(stderr, "High deviation: avg = %f, expected = %f\n", avg, expected);


  // print performance
  double flops= (double) ITER1 * ITER2 * N*2 / (t3 - t0);

   // release memory
  free(x);
  free(y);
  c( hipFree(d_x) );
  c( hipFree(d_y) );

  //c( cudaStreamDestroy(stream) );

  return flops;
}

void showDeviceProps(int dev)
{
  hipDeviceProp_t props;
  c( hipGetDeviceProperties(&props, dev) );

  printf("==== DEVICE PROPERTIES: %d ======\n", dev);
  printf(" . asyncEngineCount = %d\n", props.asyncEngineCount);
  printf("=================================\n");
}

int main(int argc, char ** argv)
{
  c( hipDeviceReset() );
  c( hipSetDeviceFlags( hipDeviceScheduleSpin ) );

  hipGetLastError(); // clear cuda errors;

  const int MINN=1<<7;
  const int MAXN=1<<17;
  const int MAXTHREADS=1<<9;
  printf("% 10s  % 10s  % 10s\n", "N", "Threads", "GFLOPS/s");
  for(int i = MINN; i <= MAXN; i*=2)
  {
    for (int j = 1; j <= MAXTHREADS; j*=2)
      printf("% 10d  % 10d  % 10f\n", i, j, measureSaxpy(i, j) * 1e-09);
  }

  return 0;
}


// vim: ft=cpp:ai
