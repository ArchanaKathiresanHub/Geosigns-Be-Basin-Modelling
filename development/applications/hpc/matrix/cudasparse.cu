#include "cudasparse.h"
#include "cudaexception.h"
#include <iostream>

#include <hipsparse.h>

#if HAVE_MPI
  #include <mpi.h>
#endif

namespace hpc
{

CudaSparse
  :: CudaSparse()
   : m_deviceCount(1)
   , m_deviceRank(0)
   , m_handle(0)
{
  CudaException :: check(   hipGetDeviceCount(&m_deviceCount) );
 
  int rank = 0 ;
  int procs = 1;

#if HAVE_MPI
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &procs);
#endif

  m_deviceRank = rank % m_deviceCount;
  CudaException :: check(   hipSetDevice( m_deviceRank ) );

  CuSparseException::check( hipsparseCreate( & m_handle ));
  CuSparseException::check( hipsparseSetPointerMode(m_handle, HIPSPARSE_POINTER_MODE_HOST) );
}

CudaSparse
  :: ~CudaSparse()
{
  hipsparseDestroy(m_handle);
}

CudaSparse &
CudaSparse
  :: instance() 
{
  static CudaSparse object;
  return object;
}

}
