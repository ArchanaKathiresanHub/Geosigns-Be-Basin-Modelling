#include "hip/hip_runtime.h"
/********************************************************************
  Simple benchmark program that measures the number of GigaFLOPS/sec
  an NVidia CUDA enabled GPU can perform while doing a SAXPY vector
  operation, i.e. it computes: y = alpha * x + y, where alpha is a
  scalar and x and y are vectors.

  The SAXPY uses cuBLAS: A CUDA accelerated BLAS library.
**/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hipblas.h>

#include "gettime.h"

void c(hipblasStatus_t errorcode)
{
  if (errorcode != HIPBLAS_STATUS_SUCCESS)
    fprintf(stderr, "A cuBLAS error occurred\n");
}

#define d(errorcode) \
  do \
  {\
    if (errorcode != hipSuccess) \
      fprintf(stderr, "A CUDA error on line %d: %s\n", __LINE__, hipGetErrorString(errorcode)); \
  } \
  while(0) 



double measureSaxpy( hipblasHandle_t handle, const int N)
{
  const long WORK = 1 << 25;
  const long ITER= WORK / N;
  float *x, *y;
  float *d_x, *d_y, *d_a;

  // allocate memory
  x = (float * ) malloc(sizeof(float)*N);
  y = (float *) malloc(sizeof(float)*N);

  d( hipMalloc(&d_x, sizeof(float)*N) );
  d( hipMalloc(&d_y, sizeof(float)*N) );
  d( hipMalloc(&d_a, sizeof(float)*1) );


  // initialize memory
  const float a = 2.0/3.0;
  for (int i = 0; i < N; ++i)
  {
    x[i] = rand() / (RAND_MAX + 1.0);
    y[i] = rand() / (RAND_MAX + 1.0);
  }

  // create a command stream
  hipStream_t stream;
  c( hipblasGetStream(handle, &stream) );

  // copy to device
  double t0 = getTime();
  c( hipblasSetVector(N, sizeof(float), x, 1, d_x, 1) );
  c( hipblasSetVector(N, sizeof(float), y, 1, d_y, 1) );
  c( hipblasSetVector(1, sizeof(float), &a, 1, d_a, 1) );

  // syncronize and start timer
  double t1 = getTime();

  // run kernel
  for (long i = 0; i < ITER; ++i)
    c( hipblasSaxpy(handle, N, d_a, d_x, 1, d_y, 1) );
  
  // copy result back
  c( hipblasGetVector(N, sizeof(float), d_y, 1, y, 1) );
  double t3 = getTime();

  // print some statistic based on x & y;
  double sum = 0.0;
  for (long i = 0 ; i < N; ++i)
    sum += y[i];
  double avg= sum / N;

  const double epsilon = 0.1;
  const double expected = (0.5 + ITER * 0.5 * a);
  if (fabs( avg - expected ) / expected > epsilon)
    fprintf(stderr, "High deviation: avg = %f, expected = %f\n", avg, expected);

  // print performance
  double flops= (double) ITER * N*2 / (t3 - t1);

  // force output of a silly statistic

  // release memory
  free(x);
  free(y);
  d( hipFree(d_x) );
  d( hipFree(d_y) );

  return flops;
}

int main(int argc, char ** argv)
{
  hipblasHandle_t handle;
  c( hipblasCreate(&handle) );
  c( hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE));

  const int MINN=1<<7;
  const int MAXN=1<<26;
  printf("#% 9s  % 10s\n", "N", "GFLOPS/s");
  for(int i = MINN; i <= MAXN; i*=2)
  {
    printf("% 10d  % 10f\n", i, measureSaxpy(handle, i) * 1e-09);
  }

  c( hipblasDestroy(handle) );

  return 0;
}


// vim: ft=cpp:cindent
