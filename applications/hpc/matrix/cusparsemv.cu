#include <iostream>
#include <fstream>
#include <cassert>

#include "petscmatrix.h"
#include "petscvector.h"
#include "cudaexception.h"
#include "gettime.h"
#include "cudaarray.h"

#include <hipsparse.h>

struct RuntimeException : hpc::BaseException< RuntimeException > {};

int main(int argc, char ** argv)
{
  using namespace hpc;

  // setup cuSPARSE
  hipsparseHandle_t cusparse ;
  hipsparseCreate(&cusparse);
  CudaException::check();

  hipsparseSetPointerMode(cusparse, HIPSPARSE_POINTER_MODE_DEVICE);
  CudaException::check();

  // read matrix
  if (argc < 2)
    throw RuntimeException() << "Not enough parameters.";

  std::ifstream input( argv[1] );

  if (!input)
    throw RuntimeException() << "Could not open file '" << argv[1] << "'.";

  PetscMatrix A = PetscMatrix::load(input);
  PetscVector b = PetscVector::load(input);
  PetscVector x = PetscVector::load(input);
  PetscVector r = PetscVector::load(input);

  input.close();

  const int dim = A.rows();
  const int nnz = A.numberNonZeros();
  const double alpha = 1.0;
  const double beta = 1.0;

  std::cout << "Matrix: " << A.rows() << " x " << A.columns() << " - " <<
    A.numberNonZeros() << " non-zeros" << std::endl;
  
  // copy matrices and vector to GPU
  CudaArray<double> d_A_NonZeros( A.nonZeros(), A.nonZeros() + nnz );
  CudaArray<int> d_A_RowOffsets( A.rowOffsets(), A.rowOffsets() + dim + 1);
  CudaArray<int> d_A_ColumnIndices( A.columnIndices(), A.columnIndices() + nnz);

  CudaArray<double> d_b_values( &b[0], &b[0] + dim );
  CudaArray<double> d_x_values( &x[0], &x[0] + dim );

  CudaArray<double> d_alpha( &alpha, &alpha + 1);
  CudaArray<double> d_beta( &beta, &beta + 1);
  
      
  hipsparseMatDescr_t matdescra;
  hipsparseCreateMatDescr(&matdescra); CudaException::check();
  hipsparseSetMatType(matdescra, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(matdescra, HIPSPARSE_INDEX_BASE_ZERO);

  // measure a multiplication with CSR matrix format
  { 
    int N = 0;
    const double duration = 10.0;
    double t0 = getTime(), t1 = 0.0;
    while (t0 + duration > (t1=getTime()))
    {
      for (int i = 0; i < 100; ++i, ++N)
      {
	hipsparseOperation_t transa = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseStatus_t result = hipsparseDcsrmv( cusparse
	    , transa
	    , dim, dim, nnz, d_alpha.devicePointer(), matdescra
	    , d_A_NonZeros.devicePointer()
	    , d_A_RowOffsets.devicePointer()
	    , d_A_ColumnIndices.devicePointer()
	    , d_b_values.devicePointer()
	    , d_beta.devicePointer(), d_x_values.devicePointer()
	    );

	switch(result)
	{
	  case HIPSPARSE_STATUS_SUCCESS:
	    break;
	  
	  case HIPSPARSE_STATUS_NOT_INITIALIZED:
	    throw RuntimeException() << "CUSPARSE library wasn't initialized";

	  case HIPSPARSE_STATUS_ALLOC_FAILED:
	    throw RuntimeException() << "Resource allocation failed in CUSPARSE library";
	  case HIPSPARSE_STATUS_INVALID_VALUE:
	    throw RuntimeException() << "Invalid parameter";
	  case HIPSPARSE_STATUS_ARCH_MISMATCH:
	    throw RuntimeException() << "Device architecture mismatch";
	  case HIPSPARSE_STATUS_MAPPING_ERROR:
	    throw RuntimeException() << "memory mapping error";
	  case HIPSPARSE_STATUS_EXECUTION_FAILED:
	    throw RuntimeException() << "GPU program failed to execute";
	  case HIPSPARSE_STATUS_INTERNAL_ERROR:
	    throw RuntimeException() << "Internal error";
	  case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
	    throw RuntimeException() << "Matrix type is not supported.";
	 }
      }
      hipDeviceSynchronize();
      CudaException::check();
    }
    std::cout << "\n=== CSR Format ===\n";
    std::cout << "Time/matrix = " << (t1-t0)/N << " seconds." << std::endl;
    double flops = 2.0 * N * A.numberNonZeros() / (t1 - t0);
    std::cout << "FLOPS = " << flops << std::endl;
  }

  // Convert to HYB
  hipsparseHybMat_t d_A_hyb;
  if ( hipsparseCreateHybMat(&d_A_hyb) != HIPSPARSE_STATUS_SUCCESS)
    throw RuntimeException() << "Error creating Hyb matrix";

  if ( hipsparseDcsr2hyb(cusparse, dim, dim, matdescra, d_A_NonZeros.devicePointer()
      , d_A_RowOffsets.devicePointer(), d_A_ColumnIndices.devicePointer()
      , d_A_hyb, 27, HIPSPARSE_HYB_PARTITION_AUTO) != HIPSPARSE_STATUS_SUCCESS)
    throw RuntimeException() << "Error converting matrix to Hyb";

  {  // Measure Hyb multiplication
    int N = 0;
    const double duration = 10.0;
    double t0 = getTime(), t1 = 0.0;
    while (t0 + duration > (t1=getTime()))
    {
      for (int i = 0; i < 100; ++i, ++N)
      {
	hipsparseOperation_t transa = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseStatus_t result = hipsparseDhybmv( cusparse
	    , transa
	    , d_alpha.devicePointer(), matdescra
	    , d_A_hyb
	    , d_b_values.devicePointer()
	    , d_beta.devicePointer(), d_x_values.devicePointer()
	    );

	switch(result)
	{
	  case HIPSPARSE_STATUS_SUCCESS:
	    break;
	  
	  case HIPSPARSE_STATUS_NOT_INITIALIZED:
	    throw RuntimeException() << "CUSPARSE library wasn't initialized";

	  case HIPSPARSE_STATUS_ALLOC_FAILED:
	    throw RuntimeException() << "Resource allocation failed in CUSPARSE library";
	  case HIPSPARSE_STATUS_INVALID_VALUE:
	    throw RuntimeException() << "Invalid parameter";
	  case HIPSPARSE_STATUS_ARCH_MISMATCH:
	    throw RuntimeException() << "Device architecture mismatch";
	  case HIPSPARSE_STATUS_MAPPING_ERROR:
	    throw RuntimeException() << "memory mapping error";
	  case HIPSPARSE_STATUS_EXECUTION_FAILED:
	    throw RuntimeException() << "GPU program failed to execute";
	  case HIPSPARSE_STATUS_INTERNAL_ERROR:
	    throw RuntimeException() << "Internal error";
	  case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
	    throw RuntimeException() << "Matrix type is not supported.";
	 }
      }
      hipDeviceSynchronize();
      CudaException::check();
    }
    std::cout << "\n=== HYB Format ===\n";
    std::cout << "Time/matrix = " << (t1-t0)/N << " seconds." << std::endl;
    double flops = 2.0 * N * A.numberNonZeros() / (t1 - t0);
    std::cout << "FLOPS = " << flops << std::endl;
  }

  



  hipsparseDestroyHybMat(d_A_hyb);
 

  hipsparseDestroyMatDescr(matdescra);
  hipsparseDestroy(cusparse);
  return 0;
}

